#define BLOCK 1024 
#define C10_WARP_SIZE 32

#include <hip/hip_runtime.h>

__device__ __forceinline__ void VectorizedAtomicAddPerBlock(
    const int64_t len, int tid, int threads_per_block, const float *in, float *out) {
  for (int i = tid; i < len; i += threads_per_block) {
    atomicAdd(&out[i], in[i]);
  }
}

extern "C"
__global__ void EmbeddingFW(float *output,
                            const float *table,
                            const float *ids,
                            const int N,
                            const int K,
                            const int D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * gridDim.x;

  while (idy < K) {
    auto id = static_cast<int64_t>(ids[idy]);
    
    float *out = output + idy * D;
    const float *tab = table + id * D;
    for (int i = idx; i < D; i += blockDim.x) {
      out[i] = tab[i];
    }
    idy += blockDim.y * gridDim.x;
  }
}

extern "C"
__global__ void EmbeddingGrad(float* table,
                              const float* output,
                              const float* ids,
                              const int N,
                              const int K,
                              const int D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * gridDim.x;

  while (idy < K) {
    auto id = static_cast<int>(ids[idy]);
    const float* out = output + idy * D;
    float* tab = table + id * D;
    //for (int i = idx; i < D; i += blockDim.x) {
      //atomicAdd(&tab[i], out[i]);
    //}
    VectorizedAtomicAddPerBlock(D, idx, blockDim.x, out, tab);
	idy += blockDim.y * gridDim.x;
  }
}

extern "C"
__global__ void embedding_backward_kernel(float* input, float* indices, float* grad_output, float* grad_weight,
  float* count, int64_t numel, int64_t stride, int padding_idx)
{
	  int idx = blockIdx.x * 4 + threadIdx.y;
	
	  // Each warp is responsible for an input into the LookupTable.
	  // If the preceding input has the same as this input, then the warp
	  // exits immediately. The warp also processes subsequent inputs with the
	  // same value.
	  //
	  // Input Warp
	  // 1     <warp 1>
	  // 1     <warp 1> (<warp 2> exits without doing any work)
	  // 5     <warp 3>
	  // 8     <warp 4>
	
	  // Number of values processed by each thread (grain size)
	  const int SZ = 4;
	
	  if (idx < numel
	      && (idx == 0 || input[idx] != input[idx - 1])
	      && input[idx] != padding_idx) {
	    do {
	      const int start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
	      const int weight_row = ((int) input[idx]) * stride;
	      const int grad_row = ((int) indices[idx]) * stride;
	      const float scale = count ? (float)1.0 / count[idx] : 1.0;
	
	      float gradient[SZ];
	      float weight[SZ];
	
	      #pragma unroll
	      for (int ii = 0; ii < SZ; ii++) {
	        int feature_dim = start_feature + ii * C10_WARP_SIZE;
	        if (feature_dim < stride) {
	          gradient[ii] = static_cast<float>(grad_output[grad_row + feature_dim]);
	          weight[ii] = static_cast<float>(grad_weight[weight_row + feature_dim]);
	        }
	      }
	
	      #pragma unroll
	      for (int ii = 0; ii < SZ; ii++) {
	        weight[ii] += gradient[ii] * scale;
	      }
	
	      #pragma unroll
	      for (int ii = 0; ii < SZ; ii++) {
	        int feature_dim = start_feature + ii * C10_WARP_SIZE;
	        if (feature_dim < stride) {
	            grad_weight[weight_row + feature_dim] = static_cast<float>(weight[ii]);
	        }
	      }
	
	      idx++;
	    } while (idx < numel && input[idx] == input[idx - 1]);
	  }
}
