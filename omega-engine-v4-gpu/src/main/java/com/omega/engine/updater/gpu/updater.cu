
#include <hip/hip_runtime.h>
#define BLOCK 1024 
#define ETA 1e-8 

extern "C"
__global__ void adam(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, int n, int batch, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    float tmp = diffW[i];
    mw[i] = beta1 * mw[i] + (1 - beta1) * tmp;
	vw[i] = beta2 * vw[i] + (1 - beta2) * tmp * tmp;
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate / batch * mhat / (sqrt(vhat) + ETA);
}

extern "C"
__global__ void adamw(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, float weight_decay, int n, int batch, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    mw[i] = beta1 * mw[i] + (1 - beta1) * diffW[i];
	vw[i] = beta2 * vw[i] + (1 - beta2) * diffW[i] * diffW[i];
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate / batch * (mhat / (sqrt(vhat) + ETA) + (batch * weight_decay * weight[i]));
}

extern "C"
__global__ void adam_bn(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, int n, int batch, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    mw[i] = beta1 * mw[i] + (1 - beta1) * diffW[i];
	vw[i] = beta2 * vw[i] + (1 - beta2) * diffW[i] * diffW[i];
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate / batch * mhat / (sqrt(vhat) + ETA);
}

extern "C"
__global__ void adamw_bn(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, float weight_decay, int n, int batch, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    mw[i] = beta1 * mw[i] + (1 - beta1) * diffW[i];
	vw[i] = beta2 * vw[i] + (1 - beta2) * diffW[i] * diffW[i];
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate / batch * (mhat / (sqrt(vhat) + ETA));
}

extern "C"
__global__ void sgd(float *diffW, float *v,float *weight,float momentum,float weight_decay,float learnRate, int n, int batch)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    float tmp = diffW[i] - (-weight_decay) * batch * weight[i];
	weight[i] = weight[i] - learnRate / batch * v[i];
	v[i] = tmp * momentum;
}

extern "C"
__global__ void sgd_bn(float *diffW, float *v,float *weight,float momentum,float weight_decay,float learnRate, int n, int batch)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
	weight[i] = weight[i] - learnRate / batch * v[i];
	v[i] = diffW[i] * momentum;
}