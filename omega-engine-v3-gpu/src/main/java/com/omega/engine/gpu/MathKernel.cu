
#include <hip/hip_runtime.h>
#define BLOCK 1024 


extern "C"
__global__ void mean_cov(float* x,float* mean,int number,int channel,int height,int width,float scale)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < channel; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		for(int n = 0;n<number;n++) {	
			for(int h = 0;h<height;h++) {
				for(int w = 0;w<width;w++) {
				
					val += x[n * channel * height * width + index * height * width + h * width + w];
	
				}
			}
		}	
		
		mean[index] = val * scale;
	}

}


extern "C"
__global__ void mean_full(float* x,float* mean,int number,int width,float scale)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < width; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		for(int n = 0;n<number;n++) {	
				
			val += x[n * width + index];
	
		}	
		
		mean[index] = val * scale;
	}

}

extern "C"
__global__ void var_cov(float* x,float* mean,float* var,int number,int channel,int height,int width,float scale)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < channel; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		float mean_val = mean[index];
		
		for(int n = 0;n<number;n++) {	
			for(int h = 0;h<height;h++) {
				for(int w = 0;w<width;w++) {
					
					float x_val = x[n * channel * height * width + index * height * width + h * width + w];
					
					val += (x_val - mean_val) * (x_val - mean_val);
	
				}
			}
		}	
		
		var[index] = val * scale;
	}

}

extern "C"
__global__ void var_full(float* x,float* mean,float* var,int number,int width,float scale)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < width; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		float mean_val = mean[index];
		
		for(int n = 0;n<number;n++) {	
				
			float x_val = x[n * width + index];
				
			val += (x_val - mean_val) * (x_val - mean_val);
	
		}	
		
		var[index] = val * scale;
	}

}


extern "C"
__global__ void std_fn(float* var,float* std,float eta,int n)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
		
		std[index] = sqrt(var[index] + eta);
		
	}

}


extern "C"
__global__ void mwa(float* mean,float* std,float* runingMean,float* runingStd,int n)
{
    
    float alpha = 0.1;
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
		
		runingMean[index] = alpha * runingMean[index] + (1 - alpha) * mean[index];
		
		runingStd[index] = alpha * runingStd[index] + (1 - alpha) * std[index];
		
	}

}


extern "C"
__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? x[index] : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        mean[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean[filter] += local[i];
        }
        mean[filter] /= spatial * batch;
    }
}


extern "C"
__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? powf((x[index] - mean[filter]), 2) : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        variance[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
    }
}