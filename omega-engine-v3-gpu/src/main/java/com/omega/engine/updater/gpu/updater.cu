
#include <hip/hip_runtime.h>
#define BLOCK 1024 
#define ETA 10e-8 


extern "C"
__global__ void adam(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, int n, int batch, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    float tmp = diffW[i] / batch;
    mw[i] = beta1 * mw[i] + (1 - beta1) * tmp;
	vw[i] = beta2 * vw[i] + (1 - beta2) * tmp * tmp;
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate * mhat / (sqrt(vhat) + ETA);
}

extern "C"
__global__ void adam_bn(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, int n, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    mw[i] = beta1 * mw[i] + (1 - beta1) * diffW[i];
	vw[i] = beta2 * vw[i] + (1 - beta2) * diffW[i] * diffW[i];
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate * mhat / (sqrt(vhat) + ETA);
}