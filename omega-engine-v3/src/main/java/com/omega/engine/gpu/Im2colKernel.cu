
#include <hip/hip_runtime.h>
extern "C"
__global__ void im2col_gpu(float *x,float *out,int N,int C,int H,int W,int kh,int kw,int stride,int oHeight,int oWidth,int oh,int ow,int kSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < oh){
   		
		int n = i / oHeight / oWidth;
				
		int startH = (i - (n * oHeight * oWidth)) / oWidth * stride;
		
		int startW = (i - (n * oHeight * oWidth)) % oWidth * stride;
	
		for(int j = 0;j<ow;j++) {
			
			int c = j / kSize;
				
			int xSize = j - (c * kSize);
			
			int xh = startH + xSize / kh;
			
			int xw = startW + xSize % kw;
			
			out[i * ow + j] = x[n * C * H * W + c * H * W + xh * W + xw];
	
		}
   		
    }
   
}

extern "C"
__global__ void im2col_gpuv2(float *x,float *out,int N,int C,int H,int W,int kh,int kw,int stride,int oHeight,int oWidth,int oh,int ow,int kSize)
{
	
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int s = blockDim.x * gridDim.x;
    
    //printf("Thread %d %d\n", index, s);
   
	for(int i = index;i < oh;i += s){
		
		int n = i / oHeight / oWidth;
				
		int startH = (i - (n * oHeight * oWidth)) / oWidth * stride;
		
		int startW = (i - (n * oHeight * oWidth)) % oWidth * stride;
	
		for(int j = 0;j<ow;j++) {
			
			int c = j / kSize;
			
			int xSize = j - (c * kSize);
			
			int xh = startH + xSize / kh;
			
			int xw = startW + xSize % kw;
			
			out[i * ow + j] = x[n * C * H * W + c * H * W + xh * W + xw];
	
		}
		
	}
	
}

extern "C"
__global__ void im2col_gpuv3(float *x,float *out,int N,int C,int H,int W,int kh,int kw,int stride,int oHeight,int oWidth,int oh,int ow,int kSize)
{
	
    int ih = blockIdx.x * blockDim.x + threadIdx.x;
    
    int iw = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(ih < oh){
    	
	    int n = ih / oHeight / oWidth;
					
		int startH = (ih - (n * oHeight * oWidth)) / oWidth * stride;
			
		int startW = (ih - (n * oHeight * oWidth)) % oWidth * stride;
    
    	if(iw < ow){
    	
			int c = iw / kSize;
			
			int xSize = iw - (c * kSize);
			
			int xh = startH + xSize / kh;
			
			int xw = startW + xSize % kw;
			
			out[ih * ow + iw] = x[n * C * H * W + c * H * W + xh * W + xw];
			
    	}
    }
    
}


extern "C"
__global__ void im2col_gpuv4(float *x,float *out,int N,int C,int H,int W,int kh,int kw,int stride,int oHeight,int oWidth,int oh,int ow,int kSize)
{
 
     int i = blockIdx.x * blockDim.x + threadIdx.x;
    
     int ih = i / ow;
    
     int iw = i % ow;
    
	 if(ih < oh){
	     
	     int n = ih / oHeight / oWidth;
	     
	     int startH = (ih - (n * oHeight * oWidth)) / oWidth * stride;
	   
	  	 int startW = (ih - (n * oHeight * oWidth)) % oWidth * stride;
	    
	     if(iw < ow){
	     
	   		int c = iw / kSize;
	   
	   		int xSize = iw - (c * kSize);
	   
	   		int xh = startH + xSize / kh;
	   
	   		int xw = startW + xSize % kw;
	   
	   		out[ih * ow + iw] = x[n * C * H * W + c * H * W + xh * W + xw];
	   
	     }
	  }
    
}

extern "C"
__global__ void im2col_gpuV5(float *x,float *out,int N,int C,int H,int W,int kh,int kw,int stride,int oHeight,int oWidth,int oh,int ow,int kSize)
{
   
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < oh; i += blockDim.x * gridDim.x) {
   		
		int n = i / oHeight / oWidth;
				
		int startH = (i - (n * oHeight * oWidth)) / oWidth * stride;
		
		int startW = (i - (n * oHeight * oWidth)) % oWidth * stride;
	
		for(int j = 0;j<ow;j++) {
			
			int c = j / kSize;
				
			int xSize = j - (c * kSize);
			
			int xh = startH + xSize / kh;
			
			int xw = startW + xSize % kw;
			
			out[i * ow + j] = x[n * C * H * W + c * H * W + xh * W + xw];
	
		}
   		
    }
   
}

extern "C"
__global__ void im2col_gpuV6(float *x,float *out,int N,int C,int H,int W,int kh,int kw,int stride,int oHeight,int oWidth,int oh,int ow,int kSize)
{
   
   const int index = blockIdx.x * blockDim.x + threadIdx.x;
   const int os = blockDim.x * gridDim.x;
   
   for(int n = 0;n<N;n++){
   
     for (int i = index; i < oh; i += os) {
   
	   const int startH = i / oWidth * stride;
		
	   const int startW = i % oWidth * stride;

	   for(int j = 0;j<ow;j++) {
			
			const int c = j / kSize;
				
			const int xSize = j - (c * kSize);
			
			const int xh = startH + xSize / kh;
			
			const int xw = startW + xSize % kw;
			
			out[n * oh * ow + i * ow + j] = x[n * C * H * W + c * H * W + xh * W + xw];
	
		}
   		
     }
   
   }
   
}

extern "C"
__global__ void im2col_gpu_kernel(float* data_im,float* data_col,int n,int height,int width,int kh,int kw,int s,int oh,int ow) {

    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
        const int h_index = index / ow;
        const int h_col = h_index % oh;
        const int w_col = index % ow;
        const int c_im = h_index / oh;
        const int c_col = c_im * kh * kw;
        const int h_offset = h_col * s;
        const int w_offset = w_col * s;
        float* data_col_ptr = data_col;
        data_col_ptr += (c_col * oh + h_col) * ow + w_col;
        const float* data_im_ptr = data_im;
        data_im_ptr += (c_im * height + h_offset) * width + w_offset;
        for (int i = 0; i < kh; ++i) {
            for (int j = 0; j < kw; ++j) {
                int h_im = h_offset + i;
                int w_im = w_offset + j;
                *data_col_ptr =
                    (h_im >= 0 && w_im >= 0 && h_im < height&& w_im < width) ?
                    data_im_ptr[i * width + j] : 0;
                data_col_ptr += oh * ow;
            }
        }
    }

}

extern "C"
__global__ void bu_im2col_gpu_kernel(
    const int n, const float* data_im,
    const int height, const int width, const int ksize,
    const int stride, const int height_col, const int width_col,
    float* data_col,
    const int data_im_size,
    const int data_col_size,
    const int batch_size)
{
    for (int batch_index = 0; batch_index < batch_size; batch_index++)
    {
        for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
            int w_out = index % width_col;
            int h_index = index / width_col;
            int h_out = h_index % height_col;
            int channel_in = h_index / height_col;
            int channel_out = channel_in * ksize * ksize;
            int h_in = h_out * stride;
            int w_in = w_out * stride;
            float* data_col_ptr = data_col;
            data_col_ptr += batch_index * data_col_size + (channel_out * height_col + h_out) * width_col + w_out;
            const float* data_im_ptr = data_im;
            data_im_ptr += batch_index * data_im_size + (channel_in * height + h_in) * width + w_in;

            for (int i = 0; i < ksize; ++i) {
                for (int j = 0; j < ksize; ++j) {
                    int h = h_in + i;
                    int w = w_in + j;
                    *data_col_ptr = (h >= 0 && w >= 0 && h < height&& w < width) ?
                        data_im_ptr[i * width + j] : 0;
                    data_col_ptr += height_col * width_col;
                }
            }

        }
    }
}

extern "C"
__global__ void test(int n,int *o)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	
}